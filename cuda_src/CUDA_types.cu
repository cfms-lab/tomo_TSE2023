#include "CUDA_types.cuh"

using namespace Tomo;

hipEvent_t start, stop;

void  cu_startTimer(void)
{
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
}

void  cu_endTimer(const char* title)
{

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	if (title != nullptr) std::cout << title << " ";
	std::cout << "time=" << milliseconds << "[��] =" << milliseconds * 1000. << "[��]" << std::endl;
	hipEventDestroy(start);
	hipEventDestroy(stop);
}

